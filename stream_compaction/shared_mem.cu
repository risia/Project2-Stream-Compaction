#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"
#include "shared_mem.h"

#define blockSize 128

// for reducing bank conflicts
#define NUM_BANKS 16
#define LOG_NUM_BANKS 4
#define CONFLICT_FREE_OFFSET(n) \
    ((n) >> NUM_BANKS + (n) >> (2 * LOG_NUM_BANKS))

namespace StreamCompaction {
    namespace SharedMem {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
		__global__ void kernScanBlockSum(int n, int* sum_buf) {

			int index = (blockDim.x * blockIdx.x) + threadIdx.x;

			int offset;
			int access;
			int a2;

			int temp;

			// Upsweep
			for (offset = 1; offset < blockSize; offset *= 2) {
				access = (2 * offset * (index + 1)) - 1;
				a2 = access - offset;
				if (access < blockSize) sum_buf[access] += sum_buf[a2];
				__syncthreads(); // avoid mem issues
			}
			if (index >= n - 1) sum_buf[index] = 0;
			__syncthreads(); // avoid mem issues

			//downsweep
			for (offset = blockSize; offset >= 1; offset /= 2) {
				access = (2 * offset * (index + 1)) - 1;
				a2 = access - offset;
				if (access < blockSize) {
					temp = sum_buf[a2]; // store left child
					sum_buf[a2] = sum_buf[access]; // swap
					sum_buf[access] += temp; // add
				}
				__syncthreads(); // avoid mem issues
			}

		}

		__global__ void kernScanDataShared(int n, int* in, int* out, int* sums) {
			// init shared mem for block, could improve latency
			__shared__ int sBuf[blockSize];

			int tx = threadIdx.x;
			int index = (blockDim.x * blockIdx.x) + tx;

			int off_tx = tx + CONFLICT_FREE_OFFSET(tx);

			// copy used vals to shared mem
			sBuf[off_tx] = (index < n) ? in[index] : 0;

			__syncthreads(); // avoid mem issues

			int offset; // step size
			int access; // shared buffer access index
			int a2;

			// Upsweep
			for (offset = 1; offset < blockSize; offset *=2) {
				access = (2 * offset * (tx + 1)) - 1;
				a2 = access - offset;
				a2 += CONFLICT_FREE_OFFSET(a2);
				access += CONFLICT_FREE_OFFSET(access);
				if (access < blockSize) sBuf[access] += sBuf[a2];
				__syncthreads(); // avoid mem issues
			}

			// prepare array for downsweep
			if (tx == blockSize - 1 + CONFLICT_FREE_OFFSET(blockSize - 1)) {
				sums[blockIdx.x] = sBuf[off_tx];
				sBuf[off_tx] = 0;
			}
			__syncthreads();
			if (index >= n - 1) sBuf[off_tx] = 0;
			__syncthreads(); // avoid mem issues

			// Downsweep (inclusive)
			// do exclusive downsweep
			int temp;

			for (offset = blockSize; offset >= 1; offset /= 2) {
				access = (2 * offset * (tx + 1)) - 1;
				a2 = access - offset;
				a2 += CONFLICT_FREE_OFFSET(a2);
				access += CONFLICT_FREE_OFFSET(access);
				if (access < blockSize) {
					temp = sBuf[a2]; // store left child
					sBuf[a2] = sBuf[access]; // swap
					sBuf[access] += temp; // add
				}
				__syncthreads(); // avoid mem issues
			}
			
			// write to dev memory
			if (index < n) {
				out[index] = sBuf[off_tx];
			}
		}

		__global__ void kernStitch(int n, int* in, int* sums) {
			int bx = blockIdx.x;
			int index = (blockDim.x * bx) + threadIdx.x;;

			if (bx == 0) return;
			if (index >= n) return;
			in[index] += sums[bx];
		}

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {

			int mod = n % blockSize;
			int size = n;

			if (mod != 0) size+= blockSize - mod;

			int num_blocks = size / blockSize;

			dim3 fullBlocksPerGrid(num_blocks);

			int* dev_out; // data to output
			int* dev_in; // input data

			int* dev_sums;

			int x;

			hipMalloc((void**)&dev_in, n * sizeof(int));
			hipMalloc((void**)&dev_out, n * sizeof(int));
			hipMalloc((void**)&dev_sums, num_blocks * sizeof(int));

			// copy input data to device
			hipMemcpy(dev_in, idata, n * sizeof(int), hipMemcpyHostToDevice);

			hipMemset(dev_out, 0, n * sizeof(int));
			checkCUDAError("initializing shared mem scan data buff fail!");

			timer().startGpuTimer();

			// scan blocks of data
			kernScanDataShared<<<fullBlocksPerGrid, blockSize>>>(n, dev_in, dev_out, dev_sums);
			checkCUDAError("shared mem scan fail!");

			

			fullBlocksPerGrid.x = (num_blocks + blockSize - 1) / blockSize;
			// scan sums from blocks
			kernScanBlockSum << <fullBlocksPerGrid, blockSize >> >(num_blocks, dev_sums);
			checkCUDAError("shared mem block scan fail!");

			fullBlocksPerGrid.x = num_blocks;
			kernStitch << <fullBlocksPerGrid, blockSize >> >(n, dev_out, dev_sums);
			checkCUDAError("shared mem scan stitch fail!");


			timer().endGpuTimer();

			// copy out data
			hipMemcpy(odata, dev_out, n * sizeof(int), hipMemcpyDeviceToHost);
			checkCUDAError("shared mem scan output copy fail!");

			for (int i = 0; i < num_blocks; i++) {
				hipMemcpy(&x, dev_sums + i, sizeof(int), hipMemcpyDeviceToHost);
				printf("Sum %i: %i\n", i, x);
			}

			hipFree(dev_out);
			hipFree(dev_in);
			hipFree(dev_sums);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {

			int* dev_map; // bool mapping
			int* dev_scan; // scanned data
			int* dev_out; // compacted data to output
			int* dev_in; // input data

			int* dev_sums;

			int mod = n % blockSize;
			int size = n;
			if (mod != 0) size += blockSize - mod;

			int num_blocks = size / blockSize;

			dim3 fullBlocksPerGrid(num_blocks);

			// allocate memory
			hipMalloc((void**)&dev_in, n * sizeof(int));
			hipMalloc((void**)&dev_map, n * sizeof(int));
			hipMalloc((void**)&dev_out, n * sizeof(int));
			hipMalloc((void**)&dev_scan, n * sizeof(int));

			hipMalloc((void**)&dev_sums, num_blocks * sizeof(int));
			checkCUDAError("shared mem compact malloc fail!");

			hipMemset(dev_scan, 0, n * sizeof(int));
			checkCUDAError("initializing shared mem scan data buff fail!");

			hipMemcpy(dev_in, idata, n * sizeof(int), hipMemcpyHostToDevice); // copy input data
			checkCUDAError("initializing w-e compact data buffs fail!");

			

            timer().startGpuTimer();
            // map
			fullBlocksPerGrid.x = ((n + blockSize - 1) / blockSize);
			StreamCompaction::Common::kernMapToBoolean << <fullBlocksPerGrid, blockSize >> >(n, dev_map, dev_in);
			checkCUDAError("w-e compact bool mapping fail!");

			// scan the map
			fullBlocksPerGrid.x = num_blocks;
			kernScanDataShared << <fullBlocksPerGrid, blockSize >> >(n, dev_map, dev_scan, dev_sums);
			checkCUDAError("shared mem scan fail!");

			int r_val;
			hipMemcpy(&r_val, dev_sums + num_blocks - 1, sizeof(int), hipMemcpyDeviceToHost);

			fullBlocksPerGrid.x = (num_blocks + blockSize - 1) / blockSize;
			// scan sums from blocks
			kernScanBlockSum << <fullBlocksPerGrid, blockSize >> >(num_blocks, dev_sums);
			checkCUDAError("shared mem block scan fail!");

			fullBlocksPerGrid.x = num_blocks;
			kernStitch << <fullBlocksPerGrid, blockSize >> >(n, dev_scan, dev_sums);
			checkCUDAError("shared mem scan stitch fail!");

			// scatter
			fullBlocksPerGrid.x = ((n + blockSize - 1) / blockSize);
			StreamCompaction::Common::kernScatter << <fullBlocksPerGrid, blockSize >> >(n, dev_out, dev_in, dev_map, dev_scan);
			checkCUDAError("shared mem compact scatter fail!");

	        timer().endGpuTimer();

			// copy output to host
			hipMemcpy(odata, dev_out, n * sizeof(int), hipMemcpyDeviceToHost);
			checkCUDAError("shared mem compact output copy fail!");

			// calc # of elements for return
			int r_val2;
			hipMemcpy(&r_val2, dev_sums + num_blocks -1, sizeof(int), hipMemcpyDeviceToHost);
			checkCUDAError("shared mem compact calc # elem fail!");

			// cleanup
			hipFree(dev_in);
			hipFree(dev_map);
			hipFree(dev_out);
			hipFree(dev_scan);
			hipFree(dev_sums);

            return r_val + r_val2;
        }
    }
}
