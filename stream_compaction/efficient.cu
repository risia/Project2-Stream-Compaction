#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

#define blockSize 256

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

		__global__ void kernScanDataUpSweep(int n, int offset1, int offset2, int* buff) {
			int index = (blockDim.x * blockIdx.x) + threadIdx.x;

			int access = index * offset2 - 1;
			if (access >= n || n < 1 || access < 0) return;

			buff[access] += buff[access - offset1];
			

		}
		__global__ void kernScanDataDownSweep(int n, int offset1, int offset2, int* buff) {
			int index = (blockDim.x * blockIdx.x) + threadIdx.x;

			int access = index * offset2 - 1;
			if (access >= n || n < 1 || access < 0) return;

			int temp = buff[access - offset1];
			buff[access - offset1] = buff[access];
			buff[access] += temp;

		}

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
			dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);

			int limit = ilog2ceil(n);
			int size = pow(2, limit);

			// allocate memory
			int* dev_buf;
			hipMalloc((void**)&dev_buf, size * sizeof(int));

			// copy input data to device
			hipMemset(dev_buf + n, 0, (size - n) * sizeof(int));
			hipMemcpy(dev_buf, idata, n * sizeof(int), hipMemcpyHostToDevice);

			timer().startGpuTimer();

			int d;
			int offset1;
			int offset2;
			// UpSweep
			
			for (d = 1; d <= limit; d++) {
				offset1 = pow(2, d - 1);
				offset2 = pow(2, d);
				kernScanDataUpSweep << <fullBlocksPerGrid, blockSize >> >(size, offset1, offset2, dev_buf);
				hipDeviceSynchronize();
			}

			// DownSweep
			hipMemset(dev_buf + n - 1, 0, (size - n + 1)* sizeof(int));
			for (d = limit; d >= 1; d--) {
				offset1 = pow(2, d - 1);
				offset2 = pow(2, d);
				kernScanDataDownSweep << <fullBlocksPerGrid, blockSize >> >(size, offset1, offset2, dev_buf);
				hipDeviceSynchronize();
			}


			timer().endGpuTimer();

			// for debugging
			//printf("Limit: %i, Size: %i, N: %i\n", limit, size, n);

			// copy output data to host
			hipMemcpy(odata, dev_buf, n * sizeof(int), hipMemcpyDeviceToHost);

			// cleanup
			hipFree(dev_buf);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {

			int* dev_map; // bool mapping
			int* dev_scan; // scanned data
			int* dev_out; // compacted data to output
			int* dev_in; // input data

			dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);

			int limit = ilog2ceil(n);
			int size = pow(2, limit);

			// allocate memory
			hipMalloc((void**)&dev_in, n * sizeof(int));
			hipMalloc((void**)&dev_map, n * sizeof(int));
			hipMalloc((void**)&dev_out, n * sizeof(int));
			hipMalloc((void**)&dev_scan, size * sizeof(int));

			hipMemcpy(dev_in, idata, n * sizeof(int), hipMemcpyHostToDevice);
			

            timer().startGpuTimer();
            // map
			StreamCompaction::Common::kernMapToBoolean << <fullBlocksPerGrid, blockSize >> >(n, dev_map, dev_in);

			hipMemcpy(dev_scan, dev_map, n * sizeof(int), hipMemcpyDeviceToDevice); // copy bool data to scan
			hipMemset(dev_scan + n, 0, (size - n) * sizeof(int)); // zero extra mem

			// scan

			int d;
			int offset1;
			int offset2;
			// UpSweep

			for (d = 1; d <= limit; d++) {
				offset1 = pow(2, d - 1);
				offset2 = pow(2, d);
				kernScanDataUpSweep << <fullBlocksPerGrid, blockSize >> >(size, offset1, offset2, dev_scan);
				hipDeviceSynchronize();
			}

			// DownSweep
			hipMemset(dev_scan + n - 1, 0, (size - n + 1) * sizeof(int)); // zero extra
			for (d = limit; d >= 1; d--) {
				offset1 = pow(2, d - 1);
				offset2 = pow(2, d);
				kernScanDataDownSweep << <fullBlocksPerGrid, blockSize >> >(size, offset1, offset2, dev_scan);
				hipDeviceSynchronize();
			}

			// scatter
			StreamCompaction::Common::kernScatter << <fullBlocksPerGrid, blockSize >> >(n, dev_out, dev_in, dev_map, dev_scan);

            timer().endGpuTimer();

			// copy output to host
			hipMemcpy(odata, dev_out, n * sizeof(int), hipMemcpyDeviceToHost);
			int map_val;
			int r_val;
			hipMemcpy(&r_val, dev_scan + n - 1, sizeof(int), hipMemcpyDeviceToHost);
			hipMemcpy(&map_val, dev_map + n - 1, sizeof(int), hipMemcpyDeviceToHost);

			if (map_val != 0) r_val++;

			// cleanup
			hipFree(dev_in);
			hipFree(dev_map);
			hipFree(dev_out);
			hipFree(dev_scan);

            return r_val;
        }
    }
}
