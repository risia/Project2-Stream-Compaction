#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

#define blockSize 64

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

		__global__ void kernScanDataUpSweep(int n, int offset1, int offset2, int* buff) {
			int index = (blockDim.x * blockIdx.x) + threadIdx.x;

			int access = index * offset2 - 1;
			if (access >= n || n < 1 || access < 0) return;

			buff[access] += buff[access - offset1];
		}


		__global__ void kernScanDataDownSweep(int n, int offset1, int offset2, int* buff) {
			int index = (blockDim.x * blockIdx.x) + threadIdx.x;

			int access = index * offset2 - 1;
			if (access >= n || n < 1 || access < 0) return;

			int temp = buff[access - offset1];
			buff[access - offset1] = buff[access];
			buff[access] += temp;
		}

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {

			int limit = ilog2ceil(n);
			int size = pow(2, limit);


			dim3 fullBlocksPerGrid((size + blockSize - 1) / blockSize);

			// allocate memory
			int* dev_buf;
			hipMalloc((void**)&dev_buf, size * sizeof(int));
			checkCUDAError("w-e scan malloc fail!");

			// copy input data to device
			hipMemset(dev_buf + n, 0, (size - n) * sizeof(int));
			hipMemcpy(dev_buf, idata, n * sizeof(int), hipMemcpyHostToDevice);
			checkCUDAError("initializing w-e scan data buff fail!");

			timer().startGpuTimer();

			int d;
			int offset1;
			int offset2;

			int threads;

			// UpSweep
			for (d = 1; d <= limit; d++) {
				offset1 = pow(2, d - 1);
				offset2 = pow(2, d);

				threads = (size / offset2);
				fullBlocksPerGrid.x = (threads / blockSize) + 1;

				kernScanDataUpSweep << <fullBlocksPerGrid, blockSize >> >(size, offset1, offset2, dev_buf);
				checkCUDAError("upsweep fail!");
			}

			// DownSweep
			hipMemset(dev_buf + n - 1, 0, (size - n + 1)* sizeof(int));
			for (d = limit; d >= 1; d--) {
				offset1 = pow(2, d - 1);
				offset2 = pow(2, d);

				threads = (size / offset2);
				fullBlocksPerGrid.x = (threads / blockSize) + 1;

				kernScanDataDownSweep << <fullBlocksPerGrid, blockSize >> >(size, offset1, offset2, dev_buf);
				checkCUDAError("downsweep fail!");
			}


			timer().endGpuTimer();

			// copy output data to host
			hipMemcpy(odata, dev_buf, n * sizeof(int), hipMemcpyDeviceToHost);
			checkCUDAError("copying output data fail!");

			// cleanup
			hipFree(dev_buf);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {

			int* dev_map; // bool mapping
			int* dev_scan; // scanned data
			int* dev_out; // compacted data to output
			int* dev_in; // input data

			int limit = ilog2ceil(n);
			int size = pow(2, limit);

			// allocate memory
			hipMalloc((void**)&dev_in, n * sizeof(int));
			hipMalloc((void**)&dev_map, n * sizeof(int));
			hipMalloc((void**)&dev_out, n * sizeof(int));
			hipMalloc((void**)&dev_scan, size * sizeof(int));
			checkCUDAError("w-e compact malloc fail!");

			hipMemset(dev_scan + n, 0, (size - n) * sizeof(int)); // zero extra mem
			hipMemcpy(dev_in, idata, n * sizeof(int), hipMemcpyHostToDevice); // copy input data
			checkCUDAError("initializing w-e compact data buffs fail!");

			dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);

            timer().startGpuTimer();
            // map
			StreamCompaction::Common::kernMapToBoolean << <fullBlocksPerGrid, blockSize >> >(n, dev_map, dev_in);
			hipMemcpy(dev_scan, dev_map, n * sizeof(int), hipMemcpyDeviceToDevice); // copy bool data to scan
			checkCUDAError("w-e compact bool mapping fail!");

			// scan

			int d;
			int offset1;
			int offset2;

			// UpSweep
			for (d = 1; d <= limit; d++) {
				offset1 = pow(2, d - 1);
				offset2 = pow(2, d);
				fullBlocksPerGrid.x = ((size / offset2) + blockSize) / blockSize;
				kernScanDataUpSweep << <fullBlocksPerGrid, blockSize >> >(size, offset1, offset2, dev_scan);
				checkCUDAError("w-e compact upsweep fail!");
			}

			// DownSweep
			hipMemset(dev_scan + n - 1, 0, (size - n + 1) * sizeof(int));
			for (d = limit; d >= 1; d--) {
				offset1 = pow(2, d - 1);
				offset2 = pow(2, d);
				fullBlocksPerGrid.x = ((size / offset2) + blockSize) / blockSize;
				kernScanDataDownSweep << <fullBlocksPerGrid, blockSize >> >(size, offset1, offset2, dev_scan);
				checkCUDAError("w-e compact downsweep fail!");
			}

			// scatter
			fullBlocksPerGrid.x = ((n + blockSize - 1) / blockSize);
			StreamCompaction::Common::kernScatter << <fullBlocksPerGrid, blockSize >> >(n, dev_out, dev_in, dev_map, dev_scan);
			checkCUDAError("w-e compact scatter fail!");

            timer().endGpuTimer();

			// copy output to host
			hipMemcpy(odata, dev_out, n * sizeof(int), hipMemcpyDeviceToHost);
			checkCUDAError("w-e compact output copy fail!");

			// calc # of elements for return
			int map_val;
			int r_val;
			hipMemcpy(&r_val, dev_scan + n - 1, sizeof(int), hipMemcpyDeviceToHost);
			hipMemcpy(&map_val, dev_map + n - 1, sizeof(int), hipMemcpyDeviceToHost);
			checkCUDAError("w-e compact calc # elem fail!");

			r_val += map_val;

			// cleanup
			hipFree(dev_in);
			hipFree(dev_map);
			hipFree(dev_out);
			hipFree(dev_scan);

            return r_val;
        }
    }
}
