#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
			thrust::host_vector<int> host_thrust_in(idata, idata + n);
			thrust::host_vector<int> host_thrust_out(odata, odata + n);

			thrust::device_vector<int> dev_thrust_in = host_thrust_in;
			thrust::device_vector<int> dev_thrust_out = host_thrust_out;


            timer().startGpuTimer();

            thrust::exclusive_scan(dev_thrust_in.begin(), dev_thrust_in.end(), dev_thrust_out.begin());

            timer().endGpuTimer();

			thrust::copy(dev_thrust_out.begin(), dev_thrust_out.end(), odata);
        }
    }
}
